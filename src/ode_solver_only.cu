#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "ode_solver_only_gpu.h"

__global__
void process_delta(int nb_signal, int degradstep, int degradrate, int ***rna_produce_protein_array,
                   int **nb_rna_produce_protein, int *nb_rna_produce,   double **protein_concentration_array,
                   double **rna_basal_concentration_array, int *nb_protein_array, int *nb_rna_array,
                   double ***rna_influence_enhancing_coef_array, double ***rna_influence_operating_coef_array,
                   int **nb_rna_influence_enhancing_coef, int  **nb_rna_influence_operating_coef,
                   double **env_concentration_array, double hill_shape, double hill_shape_n) {



  double delta = 0;

  int indiv_id = blockIdx.x;
  int prot_id = threadIdx.x;

  if (prot_id < nb_protein_array[indiv_id] - nb_signal) {
    for (int j = 0; j < degradstep; j++) {
      for (int j = 0;
           j < nb_rna_produce_protein[indiv_id][prot_id]; j++) {
        double enhancer_activity = 0;
        double operator_activity = 0;

        int rna_id = rna_produce_protein_array[indiv_id][prot_id][j];

        for (int i = 0; i <
                        nb_rna_influence_enhancing_coef[indiv_id][rna_id]; i++) {

          enhancer_activity +=
              rna_influence_enhancing_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
          operator_activity +=
              rna_influence_operating_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
        }

        double enhancer_activity_pow_n = enhancer_activity == 0 ? 0 :
                                         powf(enhancer_activity, hill_shape_n);
        double operator_activity_pow_n = operator_activity == 0 ? 0 :
                                         powf(operator_activity, hill_shape_n);
        delta += rna_basal_concentration_array[indiv_id][rna_id]
                 * (hill_shape
                    / (operator_activity_pow_n + hill_shape))
                 * (1 +
                    ((1 / rna_basal_concentration_array[indiv_id][rna_id]
                     ) -
                     1)
                    * (enhancer_activity_pow_n /
                       (enhancer_activity_pow_n + hill_shape)));
      }

      delta -=
          degradrate *
          protein_concentration_array[indiv_id][prot_id];
      delta *= 1 / (double) degradstep;

      __syncthreads();

      protein_concentration_array[indiv_id][prot_id] += delta[prot_id];
    }
  }
}
