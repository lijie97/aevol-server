#include "hip/hip_runtime.h"

#include "ode_solver_only_gpu.h"

void call_kernel_ode_cuda(int multiply_population, int max_protein,
                          int nb_signal, int degradationstep,
                          int degradation_rate) {
  process_delta<<<1024*multiply_population,max_protein>>>(nb_signal,degradationstep,degradation_rate,
      dev_rna_produce_protein_array, dev_nb_rna_produce_protein, dev_nb_rna_produce, dev_protein_concentration_array,
      dev_rna_basal_concentration_array, dev_nb_protein_array, dev_nb_rna_array,
      dev_rna_influence_enhancing_coef_array, dev_rna_influence_operating_coef_array,
      dev_nb_rna_influence_enhancing_coef, dev_nb_rna_influence_operating_coef,
      dev_env_concentration_array,hill_shape,hill_shape_n);
}


int transfert_data_to_gpu(int pop_size, int lifestep) {
  protein_concentration_array = (double**)malloc(pop_size * sizeof(double*));
  rna_basal_concentration_array = (double**)malloc(pop_size * sizeof(double*));
  rna_produce_protein_array = (int***)malloc(pop_size * sizeof(int**));
  rna_influence_enhancing_coef_array = (double***)malloc(pop_size * sizeof(double**));
  rna_influence_operating_coef_array = (double***)malloc(pop_size * sizeof(double**));

  nb_protein_array = (int*)malloc(pop_size * sizeof(int));
  nb_rna_array = (int*)malloc(pop_size * sizeof(int));
  nb_rna_produce_protein = (int**)malloc(pop_size * sizeof(int*));
  nb_rna_produce = (int*)malloc(pop_size * sizeof(int));
  nb_rna_influence_enhancing_coef = (int**) malloc(pop_size * sizeof(int*));
  nb_rna_influence_operating_coef = (int**) malloc(pop_size * sizeof(int*));
  nb_rna_influence_enhancing_coef_l1 = (int*) malloc(pop_size * sizeof(int));
  nb_rna_influence_operating_coef_l1 = (int*) malloc(pop_size * sizeof(int));

  int max_prot = 0;

  for (int i = 0; i < pop_size; i++){
    protein_concentration_array[i] = (double*)
        malloc(protein_concentration_list[i]->size() * sizeof(double));
    nb_protein_array[i] = (int) protein_concentration_list[i]->size();

    max_prot = protein_concentration_list[i]->size() > max_prot ?
               protein_concentration_list[i]->size() : max_prot;

    for (int prot_id = 0; prot_id < protein_concentration_list[i]->size(); prot_id++) {
      protein_concentration_array[i][prot_id] =
          protein_concentration_list[i]->at(prot_id);
    }

    rna_basal_concentration_array[i] = (double*)
        malloc(rna_basal_concentration_list[i]->size() * sizeof(double));
    nb_rna_array[i] = (int)rna_basal_concentration_list[i]->size();

    for (int rna_id = 0; rna_id < rna_basal_concentration_list[i]->size(); rna_id++) {
      rna_basal_concentration_array[i][rna_id] =
          rna_basal_concentration_list[i]->at(rna_id);
    }

    rna_produce_protein_array[i] = (int**)malloc(
        rna_produce_protein_list.size() * sizeof(int*));
    nb_rna_produce_protein[i] = (int*)malloc(
        rna_produce_protein_list.size() * sizeof(int));
    nb_rna_produce[i] = rna_produce_protein_list.size();

    for (int prot_id = 0; prot_id < rna_produce_protein_list.size(); prot_id++) {
      rna_produce_protein_array[i][prot_id] = (int*)malloc(
          rna_produce_protein_list[i]->at(prot_id)->size()*sizeof(int));
      nb_rna_produce_protein[i][prot_id] =
          rna_produce_protein_list[i]->at(prot_id)->size();
      for (int ix = 0; ix < rna_produce_protein_list[i]->at(prot_id)->size(); ix++) {
        rna_produce_protein_array[i][prot_id][ix] =
            rna_produce_protein_list[i]->at(prot_id)->at(ix);

      }
    }

    rna_influence_enhancing_coef_array[i] =
        (double**)malloc(rna_influence_enhancing_coef_list.size() * sizeof(double*));
    rna_influence_operating_coef_array[i] = (
        double**)malloc(rna_influence_operating_coef_list.size() * sizeof(double*));

    nb_rna_influence_enhancing_coef_l1[i] = rna_influence_enhancing_coef_list.size();
    nb_rna_influence_operating_coef_l1[i] = rna_influence_operating_coef_list.size();

    nb_rna_influence_enhancing_coef[i] =
        (int*)malloc(rna_influence_enhancing_coef_list.size() * sizeof(int));
    nb_rna_influence_operating_coef[i] =
        (int*)malloc(rna_influence_enhancing_coef_list.size() * sizeof(int));

    for (int rna_id = 0; rna_id < rna_influence_enhancing_coef_list.size(); rna_id++) {
      rna_influence_enhancing_coef_array[i][rna_id] =
          (double*)malloc(rna_influence_enhancing_coef_list[i]->at(rna_id)->size());
      nb_rna_influence_enhancing_coef[i][rna_id] =
          rna_influence_enhancing_coef_list[i]->at(rna_id)->size();

      for (int prot_id = 0; prot_id <
                            rna_influence_enhancing_coef_list[i]->at(rna_id)->size(); prot_id++) {
        rna_influence_enhancing_coef_array[i][rna_id][prot_id] =
            rna_influence_enhancing_coef_list[i]->at(rna_id)->at(prot_id);
      }
    }

    for (int rna_id = 0; rna_id < rna_influence_operating_coef_list.size(); rna_id++) {
      rna_influence_operating_coef_array[i][rna_id] =
          (double*)malloc(rna_influence_operating_coef_list[i]->at(rna_id)->size());
      nb_rna_influence_operating_coef[i][rna_id] =
          rna_influence_operating_coef_list[i]->at(rna_id)->size();

      for (int prot_id = 0; prot_id <
                            rna_influence_operating_coef_list[i]->at(rna_id)->size(); prot_id++) {
        rna_influence_operating_coef_array[i][rna_id][prot_id] =
            rna_influence_operating_coef_list[i]->at(rna_id)->at(prot_id);
      }
    }
  }

  env_concentration_array = (double**)malloc(lifestep * sizeof(double*));
  for (int i = 0; i < lifestep; i++) {
    env_concentration_array[i] = (double*)malloc(nb_signal * sizeof(double));
    for (int j=0; j < nb_signal; j++) {
      env_concentration_array[i][j] = env_concentration_list[i]->at(j);
    }
  }

  hipMalloc((void**)&dev_protein_concentration_array, pop_size * sizeof(double *));
  hipMalloc((void**)&dev_rna_basal_concentration_array, pop_size * sizeof(double *));

  hipMalloc((void**)&dev_nb_protein_array, pop_size * sizeof(int));
  hipMalloc((void**)&dev_nb_rna_array, pop_size * sizeof(int));

  hipMemcpy(dev_nb_protein_array,
             nb_protein_array, pop_size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_nb_rna_array,
             nb_rna_array, pop_size * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&dev_rna_produce_protein_array, pop_size * sizeof(int **));
  hipMalloc((void**)&dev_nb_rna_produce_protein, pop_size * sizeof(int *));
  hipMalloc((void**)&dev_nb_rna_produce, pop_size * sizeof(int));

  hipMemcpy(dev_nb_rna_produce,
             nb_rna_produce, pop_size * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&dev_rna_influence_enhancing_coef_array,
             pop_size * sizeof(double **));
  hipMalloc((void**)&dev_rna_influence_operating_coef_array,
             pop_size * sizeof(double **));

  hipMalloc((void**)&dev_nb_rna_influence_enhancing_coef,
             pop_size * sizeof(int *));
  hipMalloc((void**)&dev_nb_rna_influence_operating_coef,
             pop_size * sizeof(int *));
  hipMalloc((void**)&dev_nb_rna_influence_enhancing_coef_l1,
             pop_size * sizeof(int));
  hipMalloc((void**)&dev_nb_rna_influence_operating_coef_l1,
             pop_size * sizeof(int));

  hipMemcpy(dev_nb_rna_influence_enhancing_coef_l1,
             nb_rna_influence_enhancing_coef_l1, pop_size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_nb_rna_influence_operating_coef_l1,
             nb_rna_influence_operating_coef_l1, pop_size * sizeof(int), hipMemcpyHostToDevice);


  for (int i = 0; i < pop_size; i++){
    hipMalloc((void **)&dev_protein_concentration_array[i],
               nb_protein_array[i] * sizeof(double));
    hipMemcpy(dev_protein_concentration_array[i],
               protein_concentration_array[i],
               nb_protein_array[i] * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void **)&dev_rna_basal_concentration_array[i],
               rna_basal_concentration_list[i]->size() * sizeof(double));
    hipMemcpy(dev_rna_basal_concentration_array[i],
               rna_basal_concentration_array[i],
               nb_rna_array[i] * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void **)&dev_rna_produce_protein_array[i],
               nb_rna_produce[i] * sizeof(int*));
    hipMalloc((void **)&dev_nb_rna_produce_protein[i],
               nb_rna_produce[i] * sizeof(int));
    hipMemcpy(dev_nb_rna_produce_protein[i],
               nb_rna_produce_protein[i],
               nb_rna_produce[i] * sizeof(int), hipMemcpyHostToDevice);

    for (int prot_id = 0; prot_id < nb_rna_produce[i]; prot_id++) {
      hipMalloc((void **)&dev_rna_produce_protein_array[i][prot_id],
                 nb_rna_produce_protein[i][prot_id] * sizeof(int));
      hipMemcpy(dev_rna_produce_protein_array[i][prot_id],
                 rna_produce_protein_array[i][prot_id],
                 nb_rna_produce_protein[i][prot_id] * sizeof(int),
                 hipMemcpyHostToDevice);
    }


    hipMalloc((void**)&dev_rna_influence_enhancing_coef_array[i],
               nb_rna_influence_enhancing_coef_l1[i] * sizeof(double *));
    hipMalloc((void**)&dev_rna_influence_operating_coef_array[i],
               nb_rna_influence_operating_coef_l1[i] * sizeof(double *));

    hipMalloc((void **)&dev_nb_rna_influence_enhancing_coef[i],
               nb_rna_influence_enhancing_coef_l1[i] * sizeof(int));
    hipMalloc((void **)&dev_nb_rna_influence_operating_coef[i],
               nb_rna_influence_operating_coef_l1[i] * sizeof(int));

    hipMemcpy(dev_nb_rna_influence_enhancing_coef[i],
               nb_rna_influence_enhancing_coef[i],
               nb_rna_influence_enhancing_coef_l1[i] * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_nb_rna_influence_operating_coef[i],
               nb_rna_influence_operating_coef[i],
               nb_rna_influence_operating_coef_l1[i] * sizeof(int),
               hipMemcpyHostToDevice);

    for (int rna_id = 0; rna_id < nb_rna_influence_enhancing_coef_l1[i]; rna_id++) {
      hipMalloc((void**)&dev_rna_influence_enhancing_coef_array[i][rna_id],
                 nb_rna_influence_enhancing_coef[i][rna_id] * sizeof(double));
      hipMemcpy(dev_rna_influence_enhancing_coef_array[i][rna_id],
                 rna_influence_enhancing_coef_array[i][rna_id],
                 nb_rna_influence_enhancing_coef[i][rna_id] * sizeof(double),
                 hipMemcpyHostToDevice);
    }


    for (int rna_id = 0; rna_id < nb_rna_influence_enhancing_coef_l1[i]; rna_id++) {
      hipMalloc((void**)&dev_rna_influence_enhancing_coef_array[i][rna_id],
                 nb_rna_influence_enhancing_coef[i][rna_id] * sizeof(double));
      hipMemcpy(dev_rna_influence_enhancing_coef_array[i][rna_id],
                 rna_influence_enhancing_coef_array[i][rna_id],
                 nb_rna_influence_enhancing_coef[i][rna_id] * sizeof(double),
                 hipMemcpyHostToDevice);
    }

    for (int rna_id = 0; rna_id < nb_rna_influence_operating_coef_l1[i]; rna_id++) {
      hipMalloc((void**)&dev_rna_influence_operating_coef_array[i][rna_id],
                 nb_rna_influence_operating_coef[i][rna_id] * sizeof(double));
      hipMemcpy(dev_rna_influence_operating_coef_array[i][rna_id],
                 rna_influence_operating_coef_array[i][rna_id],
                 nb_rna_influence_operating_coef[i][rna_id] * sizeof(double),
                 hipMemcpyHostToDevice);
    }

  }


  hipMalloc((void**)&dev_env_concentration_array, lifestep * sizeof(double *));
  for (int i = 0; i < lifestep; i++) {
    hipMalloc((void**)&dev_env_concentration_array[i], nb_signal * sizeof(double));
    hipMemcpy(dev_env_concentration_array[i],
               env_concentration_array[i],
               nb_signal * sizeof(double),
               hipMemcpyHostToDevice);
  }
}

__global__
void process_delta(int nb_signal, int degradstep, int degradrate, int ***rna_produce_protein_array,
                   int **nb_rna_produce_protein, int *nb_rna_produce,   double **protein_concentration_array,
                   double **rna_basal_concentration_array, int *nb_protein_array, int *nb_rna_array,
                   double ***rna_influence_enhancing_coef_array, double ***rna_influence_operating_coef_array,
                   int **nb_rna_influence_enhancing_coef, int  **nb_rna_influence_operating_coef,
                   double **env_concentration_array, double hill_shape, double hill_shape_n) {



  double delta = 0;

  int indiv_id = blockIdx.x;
  int prot_id = threadIdx.x;

  if (prot_id < nb_protein_array[indiv_id] - nb_signal) {
    for (int j = 0; j < degradstep; j++) {
      for (int j = 0;
           j < nb_rna_produce_protein[indiv_id][prot_id]; j++) {
        double enhancer_activity = 0;
        double operator_activity = 0;

        int rna_id = rna_produce_protein_array[indiv_id][prot_id][j];

        for (int i = 0; i <
                        nb_rna_influence_enhancing_coef[indiv_id][rna_id]; i++) {

          enhancer_activity +=
              rna_influence_enhancing_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
          operator_activity +=
              rna_influence_operating_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
        }

        double enhancer_activity_pow_n = enhancer_activity == 0 ? 0 :
                                         powf(enhancer_activity, hill_shape_n);
        double operator_activity_pow_n = operator_activity == 0 ? 0 :
                                         powf(operator_activity, hill_shape_n);
        delta += rna_basal_concentration_array[indiv_id][rna_id]
                 * (hill_shape
                    / (operator_activity_pow_n + hill_shape))
                 * (1 +
                    ((1 / rna_basal_concentration_array[indiv_id][rna_id]
                     ) -
                     1)
                    * (enhancer_activity_pow_n /
                       (enhancer_activity_pow_n + hill_shape)));
      }

      delta -=
          degradrate *
          protein_concentration_array[indiv_id][prot_id];
      delta *= 1 / (double) degradstep;

      __syncthreads();

      protein_concentration_array[indiv_id][prot_id] = delta;
    }
  }
}
