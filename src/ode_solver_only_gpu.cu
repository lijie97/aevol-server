#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <unistd.h>
#include<hip/hip_runtime_api.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>

#include "ode_solver_only_gpu.h"

void call_kernel_ode_cuda(int nb_gen, int multiply_population, int max_protein,
                          int nb_signal, int degradationstep,
                          int degradation_rate, double hill_shape_n, double hill_shape) {
  for (int gen = 0; gen < nb_gen; gen++) {
    process_delta << < 1024 * multiply_population, max_protein >> >
                                                   (nb_signal, degradationstep, degradation_rate,
                                                       dev_rna_produce_protein_array, dev_nb_rna_produce_protein, dev_nb_rna_produce, dev_protein_concentration_array,
                                                       dev_rna_basal_concentration_array, dev_nb_protein_array, dev_nb_rna_array,
                                                       dev_rna_influence_enhancing_coef_array, dev_rna_influence_operating_coef_array,
                                                       dev_nb_rna_influence_enhancing_coef, dev_nb_rna_influence_operating_coef,
                                                       dev_env_concentration_array, hill_shape, hill_shape_n);
  }

  hipDeviceSynchronize();
  hipProfilerStop();
}


void call_kernel_ode_cuda_float(int nb_gen, int multiply_population, int max_protein,
                          int nb_signal, int degradationstep,
                          int degradation_rate, double hill_shape_n, double hill_shape) {
  for (int gen = 0; gen < nb_gen; gen++) {
    process_delta_float << < 1024 * multiply_population, max_protein >> >
                                                         (nb_signal, degradationstep, degradation_rate,
                                                             dev_rna_produce_protein_array, dev_nb_rna_produce_protein, dev_nb_rna_produce, f_dev_protein_concentration_array,
                                                             f_dev_rna_basal_concentration_array, dev_nb_protein_array, dev_nb_rna_array,
                                                             f_dev_rna_influence_enhancing_coef_array, f_dev_rna_influence_operating_coef_array,
                                                             dev_nb_rna_influence_enhancing_coef, dev_nb_rna_influence_operating_coef,
                                                             f_dev_env_concentration_array, (float) hill_shape, (float) hill_shape_n);
  }

  hipDeviceSynchronize();
  hipProfilerStop();
}

void call_kernel_ode_cuda_thrust(int multiply_population,
                                int nb_signal, int degradationstep,
                                int degradation_rate, double hill_shape_n, double hill_shape) {





  /*process_delta_thrust<<<1024*multiply_population,g_max_protein>>>(nb_signal,degradationstep,degradation_rate,
      g_max_protein,g_max_rna,
      r_gpu_thrust_nb_rna_produce_protein, r_gpu_thrust_rna_produce_protein,
      r_gpu_thrust_protein_concentration,
      r_gpu_thrust_rna_basal_concentration, r_gpu_thrust_nb_protein,
      r_gpu_thrust_rna_influence_enhancing_coef, r_gpu_thrust_rna_influence_operating_coef,
      r_gpu_thrust_nb_influence,
      r_gpu_thrust_environment_concentration, hill_shape, hill_shape_n);*/
}

int transfert_data_to_gpu(int pop_size, int lifestep,
         std::vector<std::vector<double>*> const &protein_concentration_list,
         std::vector<std::vector<double>*> const &rna_basal_concentration_list,
         std::vector<std::vector<std::vector<int>*>*> const &rna_produce_protein_list,
         std::vector<std::vector<std::vector<double>*>*> const &rna_influence_enhancing_coef_list,
         std::vector<std::vector<std::vector<double>*>*> const &rna_influence_operating_coef_list,
         int nb_signal,
         std::vector<std::vector<double>*> const &env_concentration_list) {
  protein_concentration_array = (double**)malloc(pop_size * sizeof(double*));
  rna_basal_concentration_array = (double**)malloc(pop_size * sizeof(double*));
  rna_produce_protein_array = (int***)malloc(pop_size * sizeof(int**));
  rna_influence_enhancing_coef_array = (double***)malloc(pop_size * sizeof(double**));
  rna_influence_operating_coef_array = (double***)malloc(pop_size * sizeof(double**));

  nb_protein_array = (int*)malloc(pop_size * sizeof(int));
  nb_rna_array = (int*)malloc(pop_size * sizeof(int));
  nb_rna_produce_protein = (int**)malloc(pop_size * sizeof(int*));
  nb_rna_produce = (int*)malloc(pop_size * sizeof(int));
  nb_rna_influence_enhancing_coef = (int**) malloc(pop_size * sizeof(int*));
  nb_rna_influence_operating_coef = (int**) malloc(pop_size * sizeof(int*));
  nb_rna_influence_enhancing_coef_l1 = (int*) malloc(pop_size * sizeof(int));
  nb_rna_influence_operating_coef_l1 = (int*) malloc(pop_size * sizeof(int));

  int max_prot = 0, min_prot = 100000000;

  for (int i = 0; i < pop_size; i++){
    protein_concentration_array[i] = (double*)
        malloc(protein_concentration_list[i]->size() * sizeof(double));
    nb_protein_array[i] = (int) protein_concentration_list[i]->size();

    max_prot = protein_concentration_list[i]->size() > max_prot ?
               protein_concentration_list[i]->size() : max_prot;

    min_prot = protein_concentration_list[i]->size() < min_prot ?
               protein_concentration_list[i]->size() : min_prot;



    for (int prot_id = 0; prot_id < protein_concentration_list[i]->size(); prot_id++) {
      protein_concentration_array[i][prot_id] =
          protein_concentration_list[i]->at(prot_id);
    }

    rna_basal_concentration_array[i] = (double*)
        malloc(rna_basal_concentration_list[i]->size() * sizeof(double));
    nb_rna_array[i] = (int)rna_basal_concentration_list[i]->size();

    for (int rna_id = 0; rna_id < rna_basal_concentration_list[i]->size(); rna_id++) {
      rna_basal_concentration_array[i][rna_id] =
          rna_basal_concentration_list[i]->at(rna_id);
    }

    rna_produce_protein_array[i] = (int**)malloc(
        rna_produce_protein_list[i]->size() * sizeof(int*));
    nb_rna_produce_protein[i] = (int*)malloc(
        rna_produce_protein_list[i]->size() * sizeof(int));
    nb_rna_produce[i] = rna_produce_protein_list[i]->size();

    for (int prot_id = 0; prot_id < rna_produce_protein_list[i]->size(); prot_id++) {
      rna_produce_protein_array[i][prot_id] = (int*)malloc(
          rna_produce_protein_list[i]->at(prot_id)->size()*sizeof(int));
      nb_rna_produce_protein[i][prot_id] =
          rna_produce_protein_list[i]->at(prot_id)->size();
      for (int ix = 0; ix < rna_produce_protein_list[i]->at(prot_id)->size(); ix++) {
        rna_produce_protein_array[i][prot_id][ix] =
            rna_produce_protein_list[i]->at(prot_id)->at(ix);

      }
    }

    rna_influence_enhancing_coef_array[i] =
        (double**)malloc(rna_influence_enhancing_coef_list[i]->size() * sizeof(double*));
    rna_influence_operating_coef_array[i] = (
        double**)malloc(rna_influence_operating_coef_list[i]->size() * sizeof(double*));

    nb_rna_influence_enhancing_coef_l1[i] = rna_influence_enhancing_coef_list[i]->size();
    nb_rna_influence_operating_coef_l1[i] = rna_influence_operating_coef_list[i]->size();

    nb_rna_influence_enhancing_coef[i] =
        (int*)malloc(rna_influence_enhancing_coef_list[i]->size() * sizeof(int));
    nb_rna_influence_operating_coef[i] =
        (int*)malloc(rna_influence_enhancing_coef_list[i]->size() * sizeof(int));

    for (int rna_id = 0; rna_id < rna_influence_enhancing_coef_list[i]->size(); rna_id++) {

      rna_influence_enhancing_coef_array[i][rna_id] =
          (double*)malloc(rna_influence_enhancing_coef_list[i]->at(rna_id)->size() * sizeof(double));

      nb_rna_influence_enhancing_coef[i][rna_id] =
          rna_influence_enhancing_coef_list[i]->at(rna_id)->size();

      for (int prot_id = 0; prot_id <
                            rna_influence_enhancing_coef_list[i]->at(rna_id)->size(); prot_id++) {
        rna_influence_enhancing_coef_array[i][rna_id][prot_id] =
            rna_influence_enhancing_coef_list[i]->at(rna_id)->at(prot_id);
      }
    }

    for (int rna_id = 0; rna_id < rna_influence_operating_coef_list[i]->size(); rna_id++) {
      rna_influence_operating_coef_array[i][rna_id] =
          (double*)malloc(rna_influence_operating_coef_list[i]->at(rna_id)->size()* sizeof(double));
      nb_rna_influence_operating_coef[i][rna_id] =
          rna_influence_operating_coef_list[i]->at(rna_id)->size();

      for (int prot_id = 0; prot_id <
                            rna_influence_operating_coef_list[i]->at(rna_id)->size(); prot_id++) {
        rna_influence_operating_coef_array[i][rna_id][prot_id] =
            rna_influence_operating_coef_list[i]->at(rna_id)->at(prot_id);
      }
    }
  }

  env_concentration_array = (double**)malloc(lifestep * sizeof(double*));
  for (int i = 0; i < lifestep; i++) {
    env_concentration_array[i] = (double*)malloc(nb_signal * sizeof(double));
    for (int j=0; j < nb_signal; j++) {
      env_concentration_array[i][j] = env_concentration_list[i]->at(j);
    }
  }

  ///// COPY TO GPU
  // env

  double **e_d = (double**)malloc(lifestep*sizeof(double*));

  for(int i=0; i<lifestep; i++) {
    hipMalloc((void**) &e_d[i],
               nb_signal * sizeof(double));
    hipMemcpy(e_d[i], env_concentration_array[i],
               nb_signal*sizeof(double),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_env_concentration_array,lifestep*sizeof(double*));
  hipMemcpy(dev_env_concentration_array,e_d,lifestep*sizeof(double*),hipMemcpyHostToDevice);

  // rna_influence_enhancing_coef_array
  double ***h_c = (double***)malloc(pop_size*sizeof(double**));

  for(int i=0; i<pop_size; i++) {
    h_c[i] = (double**) malloc(nb_rna_influence_enhancing_coef_l1[i]*sizeof(double*));
    for(int j=0; j<nb_rna_influence_enhancing_coef_l1[i]; j++) {
      hipMalloc((void**) &h_c[i][j],
                 nb_rna_influence_enhancing_coef[i][j] * sizeof(double));
      hipMemcpy(h_c[i][j], rna_influence_enhancing_coef_array[i][j],
                 nb_rna_influence_enhancing_coef[i][j]*sizeof(double),
                 hipMemcpyHostToDevice);
    }

  }

  double ***h_c1 = (double ***) malloc(pop_size*sizeof(double **));
  for (int i=0; i<pop_size; i++){
    hipMalloc((void***)&(h_c1[i]), pop_size*sizeof(double*));
    hipMemcpy(h_c1[i], h_c[i], pop_size*sizeof(double*), hipMemcpyHostToDevice);
  }

  hipMalloc((void****)&dev_rna_influence_enhancing_coef_array,pop_size*sizeof(double**));
  hipMemcpy(dev_rna_influence_enhancing_coef_array,h_c1,pop_size*sizeof(double**),hipMemcpyHostToDevice);

  // rna_influence_operating_coef_array

  h_c = (double***)malloc(pop_size*sizeof(double**));

  for(int i=0; i<pop_size; i++) {
    h_c[i] = (double**) malloc(nb_rna_influence_operating_coef_l1[i]*sizeof(double*));
    for(int j=0; j<nb_rna_influence_operating_coef_l1[i]; j++) {
      hipMalloc((void**) &h_c[i][j],
                 nb_rna_influence_operating_coef[i][j] * sizeof(double));
      hipMemcpy(h_c[i][j], rna_influence_operating_coef_array[i][j],
                 nb_rna_influence_operating_coef[i][j]*sizeof(double),
                 hipMemcpyHostToDevice);
    }

  }

  h_c1 = (double ***) malloc(pop_size*sizeof(double **));
  for (int i=0; i<pop_size; i++){
    hipMalloc((void***)&(h_c1[i]), pop_size*sizeof(double*));
    hipMemcpy(h_c1[i], h_c[i], pop_size*sizeof(double*), hipMemcpyHostToDevice);
  }

  hipMalloc((void****)&dev_rna_influence_operating_coef_array,pop_size*sizeof(double**));
  hipMemcpy(dev_rna_influence_operating_coef_array,h_c1,pop_size*sizeof(double**),hipMemcpyHostToDevice);
  
  // protein_concentration_array (double**)malloc(pop_size * sizeof(double*));
  double **h_d = (double**)malloc(pop_size*sizeof(double*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &h_d[i],
                 nb_protein_array[i] * sizeof(double));
    hipMemcpy(h_d[i], protein_concentration_array[i],
                 nb_protein_array[i]*sizeof(double),
                 hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_protein_concentration_array,pop_size*sizeof(double*));
  hipMemcpy(dev_protein_concentration_array,h_d,pop_size*sizeof(double*),hipMemcpyHostToDevice);

  // rna_basal_concentration_array
  h_d = (double**)malloc(pop_size*sizeof(double*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &h_d[i],
               nb_rna_array[i] * sizeof(double));
    hipMemcpy(h_d[i], rna_basal_concentration_array[i],
               nb_rna_array[i]*sizeof(double),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_rna_basal_concentration_array,pop_size*sizeof(double*));
  hipMemcpy(dev_rna_basal_concentration_array,h_d,pop_size*sizeof(double*),hipMemcpyHostToDevice);

  // rna_produce_protein_array

  int ***i_c = (int***)malloc(pop_size*sizeof(int**));

  for(int i=0; i<pop_size; i++) {
    i_c[i] = (int**) malloc(nb_rna_produce[i]*sizeof(int*));
    for(int j=0; j<nb_rna_produce[i]; j++) {
      hipMalloc((void**) &i_c[i][j],
                 nb_rna_produce_protein[i][j] * sizeof(int));
      hipMemcpy(i_c[i][j], rna_produce_protein_array[i][j],
                 nb_rna_produce_protein[i][j]*sizeof(int),
                 hipMemcpyHostToDevice);
    }

  }

  int ***i_c1 = (int ***) malloc(pop_size*sizeof(int **));
  for (int i=0; i<pop_size; i++){
    hipMalloc((void***)&(i_c1[i]), pop_size*sizeof(int*));
    hipMemcpy(i_c1[i], i_c[i], pop_size*sizeof(int*), hipMemcpyHostToDevice);
  }

  hipMalloc((void****)&dev_rna_produce_protein_array,pop_size*sizeof(int**));
  hipMemcpy(dev_rna_produce_protein_array,i_c1,pop_size*sizeof(int**),hipMemcpyHostToDevice);

  // nb_protein_array
  hipMalloc((void**)&dev_nb_protein_array, pop_size * sizeof(int));
  hipMemcpy(dev_nb_protein_array,
             nb_protein_array, pop_size * sizeof(int), hipMemcpyHostToDevice);

  // nb_rna_array
  hipMalloc((void**)&dev_nb_rna_array, pop_size * sizeof(int));
  hipMemcpy(dev_nb_rna_array,
             nb_rna_array, pop_size * sizeof(int), hipMemcpyHostToDevice);

  // nb_rna_produce_protein
  int **i_d = (int**)malloc(pop_size*sizeof(int*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &i_d[i],
               nb_rna_produce[i] * sizeof(int));
    hipMemcpy(i_d[i], nb_rna_produce_protein[i],
               nb_rna_produce[i]*sizeof(int),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_nb_rna_produce_protein,pop_size*sizeof(int*));
  hipMemcpy(dev_nb_rna_produce_protein,i_d,pop_size*sizeof(int*),hipMemcpyHostToDevice);

  // nb_rna_produce
  hipMalloc((void**)&dev_nb_rna_produce, pop_size * sizeof(int));

  hipMemcpy(dev_nb_rna_produce,
             nb_rna_produce, pop_size * sizeof(int), hipMemcpyHostToDevice);

  // nb_rna_influence_enhancing_coef

  i_d = (int**)malloc(pop_size*sizeof(int*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &i_d[i],
               nb_rna_influence_enhancing_coef_l1[i] * sizeof(int));
    hipMemcpy(i_d[i], nb_rna_influence_enhancing_coef[i],
               nb_rna_influence_enhancing_coef_l1[i]*sizeof(int),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_nb_rna_influence_enhancing_coef,pop_size*sizeof(int*));
  hipMemcpy(dev_nb_rna_influence_enhancing_coef,i_d,pop_size*sizeof(int*),hipMemcpyHostToDevice);

  // nb_rna_influence_operating_coef

  i_d = (int**)malloc(pop_size*sizeof(int*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &i_d[i],
               nb_rna_influence_operating_coef_l1[i] * sizeof(int));
    hipMemcpy(i_d[i], nb_rna_influence_operating_coef[i],
               nb_rna_influence_operating_coef_l1[i]*sizeof(int),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_nb_rna_influence_operating_coef,pop_size*sizeof(int*));
  hipMemcpy(dev_nb_rna_influence_operating_coef,i_d,pop_size*sizeof(int*),hipMemcpyHostToDevice);

  // nb_rna_influence_enhancing_coef_l1
  hipMalloc((void**)&dev_nb_rna_influence_enhancing_coef_l1,
             pop_size * sizeof(int));
  hipMemcpy(dev_nb_rna_influence_enhancing_coef_l1,
             nb_rna_influence_enhancing_coef_l1, pop_size * sizeof(int), hipMemcpyHostToDevice);

  // nb_rna_influence_operating_coef_l1
  hipMalloc((void**)&dev_nb_rna_influence_operating_coef_l1,
             pop_size * sizeof(int));
  hipMemcpy(dev_nb_rna_influence_operating_coef_l1,
             nb_rna_influence_operating_coef_l1, pop_size * sizeof(int), hipMemcpyHostToDevice);
  ////

  return max_prot;
}



int transfert_data_to_gpu_float(int pop_size, int lifestep,
                          std::vector<std::vector<double>*> const &protein_concentration_list,
                          std::vector<std::vector<double>*> const &rna_basal_concentration_list,
                          std::vector<std::vector<std::vector<int>*>*> const &rna_produce_protein_list,
                          std::vector<std::vector<std::vector<double>*>*> const &rna_influence_enhancing_coef_list,
                          std::vector<std::vector<std::vector<double>*>*> const &rna_influence_operating_coef_list,
                          int nb_signal,
                          std::vector<std::vector<double>*> const &env_concentration_list) {
  f_protein_concentration_array = (float**)malloc(pop_size * sizeof(float*));
  f_rna_basal_concentration_array = (float**)malloc(pop_size * sizeof(float*));
  rna_produce_protein_array = (int***)malloc(pop_size * sizeof(int**));
  f_rna_influence_enhancing_coef_array = (float***)malloc(pop_size * sizeof(float**));
  f_rna_influence_operating_coef_array = (float***)malloc(pop_size * sizeof(float**));

  nb_protein_array = (int*)malloc(pop_size * sizeof(int));
  nb_rna_array = (int*)malloc(pop_size * sizeof(int));
  nb_rna_produce_protein = (int**)malloc(pop_size * sizeof(int*));
  nb_rna_produce = (int*)malloc(pop_size * sizeof(int));
  nb_rna_influence_enhancing_coef = (int**) malloc(pop_size * sizeof(int*));
  nb_rna_influence_operating_coef = (int**) malloc(pop_size * sizeof(int*));
  nb_rna_influence_enhancing_coef_l1 = (int*) malloc(pop_size * sizeof(int));
  nb_rna_influence_operating_coef_l1 = (int*) malloc(pop_size * sizeof(int));

  int max_prot = 0, min_prot = 100000000;

  for (int i = 0; i < pop_size; i++){
    f_protein_concentration_array[i] = (float*)
        malloc(protein_concentration_list[i]->size() * sizeof(float));
    nb_protein_array[i] = (int) protein_concentration_list[i]->size();

    max_prot = protein_concentration_list[i]->size() > max_prot ?
               protein_concentration_list[i]->size() : max_prot;

    min_prot = protein_concentration_list[i]->size() < min_prot ?
               protein_concentration_list[i]->size() : min_prot;



    for (int prot_id = 0; prot_id < protein_concentration_list[i]->size(); prot_id++) {
      f_protein_concentration_array[i][prot_id] = (float)
          protein_concentration_list[i]->at(prot_id);
    }

    f_rna_basal_concentration_array[i] = (float*)
        malloc(rna_basal_concentration_list[i]->size() * sizeof(float));
    nb_rna_array[i] = (int)rna_basal_concentration_list[i]->size();

    for (int rna_id = 0; rna_id < rna_basal_concentration_list[i]->size(); rna_id++) {
      f_rna_basal_concentration_array[i][rna_id] = (float)
          rna_basal_concentration_list[i]->at(rna_id);
    }

    rna_produce_protein_array[i] = (int**)malloc(
        rna_produce_protein_list[i]->size() * sizeof(int*));
    nb_rna_produce_protein[i] = (int*)malloc(
        rna_produce_protein_list[i]->size() * sizeof(int));
    nb_rna_produce[i] = rna_produce_protein_list[i]->size();

    for (int prot_id = 0; prot_id < rna_produce_protein_list[i]->size(); prot_id++) {
      rna_produce_protein_array[i][prot_id] = (int*)malloc(
          rna_produce_protein_list[i]->at(prot_id)->size()*sizeof(int));
      nb_rna_produce_protein[i][prot_id] =
          rna_produce_protein_list[i]->at(prot_id)->size();
      for (int ix = 0; ix < rna_produce_protein_list[i]->at(prot_id)->size(); ix++) {
        rna_produce_protein_array[i][prot_id][ix] =
            rna_produce_protein_list[i]->at(prot_id)->at(ix);

      }
    }

    f_rna_influence_enhancing_coef_array[i] =
        (float**)malloc(rna_influence_enhancing_coef_list[i]->size() * sizeof(float*));
    f_rna_influence_operating_coef_array[i] = (
        float**)malloc(rna_influence_operating_coef_list[i]->size() * sizeof(float*));

    nb_rna_influence_enhancing_coef_l1[i] = rna_influence_enhancing_coef_list[i]->size();
    nb_rna_influence_operating_coef_l1[i] = rna_influence_operating_coef_list[i]->size();

    nb_rna_influence_enhancing_coef[i] =
        (int*)malloc(rna_influence_enhancing_coef_list[i]->size() * sizeof(int));
    nb_rna_influence_operating_coef[i] =
        (int*)malloc(rna_influence_enhancing_coef_list[i]->size() * sizeof(int));

    for (int rna_id = 0; rna_id < rna_influence_enhancing_coef_list[i]->size(); rna_id++) {

      f_rna_influence_enhancing_coef_array[i][rna_id] =
          (float*)malloc(rna_influence_enhancing_coef_list[i]->at(rna_id)->size() * sizeof(float));

      nb_rna_influence_enhancing_coef[i][rna_id] =
          rna_influence_enhancing_coef_list[i]->at(rna_id)->size();

      for (int prot_id = 0; prot_id <
                            rna_influence_enhancing_coef_list[i]->at(rna_id)->size(); prot_id++) {
        f_rna_influence_enhancing_coef_array[i][rna_id][prot_id] = (float)
            rna_influence_enhancing_coef_list[i]->at(rna_id)->at(prot_id);
      }
    }

    for (int rna_id = 0; rna_id < rna_influence_operating_coef_list[i]->size(); rna_id++) {
      f_rna_influence_operating_coef_array[i][rna_id] =
          (float*)malloc(rna_influence_operating_coef_list[i]->at(rna_id)->size()* sizeof(float));
      nb_rna_influence_operating_coef[i][rna_id] =
          rna_influence_operating_coef_list[i]->at(rna_id)->size();

      for (int prot_id = 0; prot_id <
                            rna_influence_operating_coef_list[i]->at(rna_id)->size(); prot_id++) {
        f_rna_influence_operating_coef_array[i][rna_id][prot_id] = (float)
            rna_influence_operating_coef_list[i]->at(rna_id)->at(prot_id);
      }
    }
  }

  f_env_concentration_array = (float**)malloc(lifestep * sizeof(float*));
  for (int i = 0; i < lifestep; i++) {
    f_env_concentration_array[i] = (float*)malloc(nb_signal * sizeof(float));
    for (int j=0; j < nb_signal; j++) {
      f_env_concentration_array[i][j] = (float) env_concentration_list[i]->at(j);
    }
  }

  ///// COPY TO GPU
  // env

  float **e_d = (float**)malloc(lifestep*sizeof(float*));

  for(int i=0; i<lifestep; i++) {
    hipMalloc((void**) &e_d[i],
               nb_signal * sizeof(float));
    hipMemcpy(e_d[i], f_env_concentration_array[i],
               nb_signal*sizeof(float),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&f_dev_env_concentration_array,lifestep*sizeof(float*));
  hipMemcpy(f_dev_env_concentration_array,e_d,lifestep*sizeof(float*),hipMemcpyHostToDevice);

  // rna_influence_enhancing_coef_array
  float ***h_c = (float***)malloc(pop_size*sizeof(float**));

  for(int i=0; i<pop_size; i++) {
    h_c[i] = (float**) malloc(nb_rna_influence_enhancing_coef_l1[i]*sizeof(float*));
    for(int j=0; j<nb_rna_influence_enhancing_coef_l1[i]; j++) {
      hipMalloc((void**) &h_c[i][j],
                 nb_rna_influence_enhancing_coef[i][j] * sizeof(float));
      hipMemcpy(h_c[i][j], f_rna_influence_enhancing_coef_array[i][j],
                 nb_rna_influence_enhancing_coef[i][j]*sizeof(float),
                 hipMemcpyHostToDevice);
    }

  }

  float ***h_c1 = (float ***) malloc(pop_size*sizeof(float **));
  for (int i=0; i<pop_size; i++){
    hipMalloc((void***)&(h_c1[i]), pop_size*sizeof(float*));
    hipMemcpy(h_c1[i], h_c[i], pop_size*sizeof(float*), hipMemcpyHostToDevice);
  }

  hipMalloc((void****)&f_dev_rna_influence_enhancing_coef_array,pop_size*sizeof(float**));
  hipMemcpy(f_dev_rna_influence_enhancing_coef_array,h_c1,pop_size*sizeof(float**),hipMemcpyHostToDevice);

  // rna_influence_operating_coef_array

  h_c = (float***)malloc(pop_size*sizeof(float**));

  for(int i=0; i<pop_size; i++) {
    h_c[i] = (float**) malloc(nb_rna_influence_operating_coef_l1[i]*sizeof(float*));
    for(int j=0; j<nb_rna_influence_operating_coef_l1[i]; j++) {
      hipMalloc((void**) &h_c[i][j],
                 nb_rna_influence_operating_coef[i][j] * sizeof(float));
      hipMemcpy(h_c[i][j], f_rna_influence_operating_coef_array[i][j],
                 nb_rna_influence_operating_coef[i][j]*sizeof(float),
                 hipMemcpyHostToDevice);
    }

  }

  h_c1 = (float ***) malloc(pop_size*sizeof(float **));
  for (int i=0; i<pop_size; i++){
    hipMalloc((void***)&(h_c1[i]), pop_size*sizeof(float*));
    hipMemcpy(h_c1[i], h_c[i], pop_size*sizeof(float*), hipMemcpyHostToDevice);
  }

  hipMalloc((void****)&f_dev_rna_influence_operating_coef_array,pop_size*sizeof(float**));
  hipMemcpy(f_dev_rna_influence_operating_coef_array,h_c1,pop_size*sizeof(float**),hipMemcpyHostToDevice);

  // protein_concentration_array (double**)malloc(pop_size * sizeof(double*));
  float **h_d = (float**)malloc(pop_size*sizeof(float*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &h_d[i],
               nb_protein_array[i] * sizeof(float));
    hipMemcpy(h_d[i], f_protein_concentration_array[i],
               nb_protein_array[i]*sizeof(float),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&f_dev_protein_concentration_array,pop_size*sizeof(float*));
  hipMemcpy(f_dev_protein_concentration_array,h_d,pop_size*sizeof(float*),hipMemcpyHostToDevice);

  // rna_basal_concentration_array
  h_d = (float**)malloc(pop_size*sizeof(float*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &h_d[i],
               nb_rna_array[i] * sizeof(float));
    hipMemcpy(h_d[i], f_rna_basal_concentration_array[i],
               nb_rna_array[i]*sizeof(float),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&f_dev_rna_basal_concentration_array,pop_size*sizeof(float*));
  hipMemcpy(f_dev_rna_basal_concentration_array,h_d,pop_size*sizeof(float*),hipMemcpyHostToDevice);

  // rna_produce_protein_array

  int ***i_c = (int***)malloc(pop_size*sizeof(int**));

  for(int i=0; i<pop_size; i++) {
    i_c[i] = (int**) malloc(nb_rna_produce[i]*sizeof(int*));
    for(int j=0; j<nb_rna_produce[i]; j++) {
      hipMalloc((void**) &i_c[i][j],
                 nb_rna_produce_protein[i][j] * sizeof(int));
      hipMemcpy(i_c[i][j], rna_produce_protein_array[i][j],
                 nb_rna_produce_protein[i][j]*sizeof(int),
                 hipMemcpyHostToDevice);
    }

  }

  int ***i_c1 = (int ***) malloc(pop_size*sizeof(int **));
  for (int i=0; i<pop_size; i++){
    hipMalloc((void***)&(i_c1[i]), pop_size*sizeof(int*));
    hipMemcpy(i_c1[i], i_c[i], pop_size*sizeof(int*), hipMemcpyHostToDevice);
  }

  hipMalloc((void****)&dev_rna_produce_protein_array,pop_size*sizeof(int**));
  hipMemcpy(dev_rna_produce_protein_array,i_c1,pop_size*sizeof(int**),hipMemcpyHostToDevice);

  // nb_protein_array
  hipMalloc((void**)&dev_nb_protein_array, pop_size * sizeof(int));
  hipMemcpy(dev_nb_protein_array,
             nb_protein_array, pop_size * sizeof(int), hipMemcpyHostToDevice);

  // nb_rna_array
  hipMalloc((void**)&dev_nb_rna_array, pop_size * sizeof(int));
  hipMemcpy(dev_nb_rna_array,
             nb_rna_array, pop_size * sizeof(int), hipMemcpyHostToDevice);

  // nb_rna_produce_protein
  int **i_d = (int**)malloc(pop_size*sizeof(int*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &i_d[i],
               nb_rna_produce[i] * sizeof(int));
    hipMemcpy(i_d[i], nb_rna_produce_protein[i],
               nb_rna_produce[i]*sizeof(int),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_nb_rna_produce_protein,pop_size*sizeof(int*));
  hipMemcpy(dev_nb_rna_produce_protein,i_d,pop_size*sizeof(int*),hipMemcpyHostToDevice);

  // nb_rna_produce
  hipMalloc((void**)&dev_nb_rna_produce, pop_size * sizeof(int));

  hipMemcpy(dev_nb_rna_produce,
             nb_rna_produce, pop_size * sizeof(int), hipMemcpyHostToDevice);

  // nb_rna_influence_enhancing_coef

  i_d = (int**)malloc(pop_size*sizeof(int*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &i_d[i],
               nb_rna_influence_enhancing_coef_l1[i] * sizeof(int));
    hipMemcpy(i_d[i], nb_rna_influence_enhancing_coef[i],
               nb_rna_influence_enhancing_coef_l1[i]*sizeof(int),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_nb_rna_influence_enhancing_coef,pop_size*sizeof(int*));
  hipMemcpy(dev_nb_rna_influence_enhancing_coef,i_d,pop_size*sizeof(int*),hipMemcpyHostToDevice);

  // nb_rna_influence_operating_coef

  i_d = (int**)malloc(pop_size*sizeof(int*));

  for(int i=0; i<pop_size; i++) {
    hipMalloc((void**) &i_d[i],
               nb_rna_influence_operating_coef_l1[i] * sizeof(int));
    hipMemcpy(i_d[i], nb_rna_influence_operating_coef[i],
               nb_rna_influence_operating_coef_l1[i]*sizeof(int),
               hipMemcpyHostToDevice);
  }

  hipMalloc((void***)&dev_nb_rna_influence_operating_coef,pop_size*sizeof(int*));
  hipMemcpy(dev_nb_rna_influence_operating_coef,i_d,pop_size*sizeof(int*),hipMemcpyHostToDevice);

  // nb_rna_influence_enhancing_coef_l1
  hipMalloc((void**)&dev_nb_rna_influence_enhancing_coef_l1,
             pop_size * sizeof(int));
  hipMemcpy(dev_nb_rna_influence_enhancing_coef_l1,
             nb_rna_influence_enhancing_coef_l1, pop_size * sizeof(int), hipMemcpyHostToDevice);

  // nb_rna_influence_operating_coef_l1
  hipMalloc((void**)&dev_nb_rna_influence_operating_coef_l1,
             pop_size * sizeof(int));
  hipMemcpy(dev_nb_rna_influence_operating_coef_l1,
             nb_rna_influence_operating_coef_l1, pop_size * sizeof(int), hipMemcpyHostToDevice);
  ////

  return max_prot;
}


int transfert_data_to_gpu_dense(int pop_size, int lifestep,
                          std::vector<std::vector<double>*> const &protein_concentration_list,
                          std::vector<std::vector<double>*> const &rna_basal_concentration_list,
                          std::vector<std::vector<std::vector<int>*>*> const &rna_produce_protein_list,
                          std::vector<std::vector<std::vector<double>*>*> const &rna_influence_enhancing_coef_list,
                          std::vector<std::vector<std::vector<double>*>*> const &rna_influence_operating_coef_list,
                          int nb_signal,
                          std::vector<std::vector<double>*> const &env_concentration_list) {

  int max_prot = 0, max_rna = 0;

  /* Computing max rna and protein */
  for (int i = 0; i < pop_size; i++) {
    max_prot = protein_concentration_list[i]->size() > max_prot ?
               protein_concentration_list[i]->size() : max_prot;
    max_rna  = rna_basal_concentration_list[i]->size() > max_rna ?
               rna_basal_concentration_list[i]->size() : max_rna;
  }

  /* Storing as dense tab */

  protein_tab = (double*)malloc(pop_size * max_prot * sizeof(double));
  memset (protein_tab, -1, pop_size * max_prot * sizeof(double));


  rna_basal_tab = (double*)malloc(pop_size * max_rna * sizeof(double));
  memset (rna_basal_tab, -1, pop_size * max_rna * sizeof(double));

  rna_produce_protein_tab = (int*)malloc(pop_size * max_prot * max_rna * sizeof(int));
  memset (rna_produce_protein_tab, -1, pop_size * max_prot * max_rna * sizeof(int));


  rna_influence_enhancing_tab = (double*)malloc(pop_size * max_prot * max_rna * sizeof(double));
  memset (rna_influence_enhancing_tab, 0, pop_size * max_prot * max_rna * sizeof(double));

  rna_influence_operating_tab = (double*)malloc(pop_size * max_prot * max_rna * sizeof(double));
  memset (rna_influence_operating_tab, 0, pop_size * max_prot * max_rna * sizeof(double));

  for (int i = 0; i < pop_size; i++) {
    for (int prot_id = 0; prot_id < protein_concentration_list[i]->size(); prot_id++) {
      protein_tab[i*max_prot+prot_id] =
          protein_concentration_list[i]->at(prot_id);
    }

    for (int rna_id = 0; rna_id < rna_basal_concentration_list[i]->size(); rna_id++) {
      rna_basal_tab[i*max_rna+rna_id] =
          rna_basal_concentration_list[i]->at(rna_id);
    }

    for (int prot_id = 0; prot_id < rna_produce_protein_list[i]->size(); prot_id++) {
      for (int ix = 0; ix < rna_produce_protein_list[i]->at(prot_id)->size(); ix++) {
        rna_produce_protein_tab[i*max_prot+prot_id*max_rna+ix] =
            rna_produce_protein_list[i]->at(prot_id)->at(ix);

      }
    }

    for (int rna_id = 0; rna_id < rna_influence_enhancing_coef_list[i]->size(); rna_id++) {
      for (int prot_id = 0; prot_id <
                            rna_influence_enhancing_coef_list[i]->at(rna_id)->size(); prot_id++) {
        rna_influence_enhancing_tab[i*max_rna+rna_id*max_prot+prot_id] =
            rna_influence_enhancing_coef_list[i]->at(rna_id)->at(prot_id);
      }
    }

    for (int rna_id = 0; rna_id < rna_influence_operating_coef_list[i]->size(); rna_id++) {
      for (int prot_id = 0; prot_id <
                            rna_influence_operating_coef_list[i]->at(rna_id)->size(); prot_id++) {
        rna_influence_operating_tab[i*max_rna+rna_id*max_prot+prot_id] =
            rna_influence_operating_coef_list[i]->at(rna_id)->at(prot_id);
      }
    }
  }

  env_concentration_tab = (double*)malloc(lifestep * nb_signal * sizeof(double));
  for (int i = 0; i < lifestep; i++) {
    for (int j=0; j < nb_signal; j++) {
      env_concentration_tab[i*nb_signal+j] = env_concentration_list[i]->at(j);
    }
  }

  ///// COPY TO GPU
  hipMallocPitch(&dev_protein_tab, &pitch_protein_tab, sizeof(double)*max_prot, pop_size);
  hipMemcpy2D(dev_protein_tab,pitch_protein_tab,protein_tab,
               sizeof(double)*max_prot,sizeof(double)*max_prot,pop_size,
               hipMemcpyHostToDevice);

  hipMallocPitch(&dev_rna_basal_tab, &pitch_rna_basal_tab, sizeof(double)*max_rna, pop_size);
  hipMemcpy2D(dev_rna_basal_tab,pitch_rna_basal_tab,rna_basal_tab,
               sizeof(double)*max_rna,sizeof(double)*max_rna,pop_size,
               hipMemcpyHostToDevice);

  // Allocate 3D memory on the device
  hipExtent volumeSizeBytes_rna_produce =
      make_hipExtent(sizeof(int) * max_rna, max_prot, pop_size);

  hipPitchedPtr devicePitchedPointer_rna_produce;

  hipMalloc3D(&devicePitchedPointer_rna_produce, volumeSizeBytes_rna_produce);

  hipMemcpy3DParms p_rna_produce = { 0 };

  p_rna_produce.srcPtr.ptr = rna_produce_protein_tab;
  p_rna_produce.srcPtr.pitch = max_rna * sizeof(int);
  p_rna_produce.srcPtr.xsize = max_rna;
  p_rna_produce.srcPtr.ysize = max_prot;
  p_rna_produce.dstPtr.ptr = devicePitchedPointer_rna_produce.ptr;
  p_rna_produce.dstPtr.pitch = devicePitchedPointer_rna_produce.pitch;
  p_rna_produce.dstPtr.xsize = max_rna;
  p_rna_produce.dstPtr.ysize = max_prot;
  p_rna_produce.extent.width = max_rna * sizeof(int);
  p_rna_produce.extent.height = max_prot;
  p_rna_produce.extent.depth = pop_size;
  p_rna_produce.kind = hipMemcpyHostToDevice;

  hipMemcpy3D(&p_rna_produce);


  hipExtent volumeSizeBytes_influence_enhancing =
      make_hipExtent(sizeof(double) * max_prot, max_rna, pop_size);

  hipPitchedPtr devicePitchedPointer_influence_enhancing;

  hipMalloc3D(&devicePitchedPointer_influence_enhancing, volumeSizeBytes_influence_enhancing);

  hipMemcpy3DParms p_influence_enhancing = { 0 };

  p_influence_enhancing.srcPtr.ptr = rna_influence_enhancing_tab;
  p_influence_enhancing.srcPtr.pitch = max_prot * sizeof(double);
  p_influence_enhancing.srcPtr.xsize = max_prot;
  p_influence_enhancing.srcPtr.ysize = max_rna;
  p_influence_enhancing.dstPtr.ptr = devicePitchedPointer_influence_enhancing.ptr;
  p_influence_enhancing.dstPtr.pitch = devicePitchedPointer_influence_enhancing.pitch;
  p_influence_enhancing.dstPtr.xsize = max_prot;
  p_influence_enhancing.dstPtr.ysize = max_rna;
  p_influence_enhancing.extent.width = max_prot * sizeof(double);
  p_influence_enhancing.extent.height = max_rna;
  p_influence_enhancing.extent.depth = pop_size;
  p_influence_enhancing.kind = hipMemcpyHostToDevice;

  hipMemcpy3D(&p_influence_enhancing);


  hipExtent volumeSizeBytes_influence_operating =
      make_hipExtent(sizeof(double) * max_prot, max_rna, pop_size);

  hipPitchedPtr devicePitchedPointer_influence_operating;

  hipMalloc3D(&devicePitchedPointer_influence_operating, volumeSizeBytes_influence_operating);

  hipMemcpy3DParms p_influence_operating = { 0 };

  p_influence_operating.srcPtr.ptr = rna_influence_operating_tab;
  p_influence_operating.srcPtr.pitch = max_prot * sizeof(double);
  p_influence_operating.srcPtr.xsize = max_prot;
  p_influence_operating.srcPtr.ysize = max_rna;
  p_influence_operating.dstPtr.ptr = devicePitchedPointer_influence_enhancing.ptr;
  p_influence_operating.dstPtr.pitch = devicePitchedPointer_influence_enhancing.pitch;
  p_influence_operating.dstPtr.xsize = max_prot;
  p_influence_operating.dstPtr.ysize = max_rna;
  p_influence_operating.extent.width = max_prot * sizeof(double);
  p_influence_operating.extent.height = max_rna;
  p_influence_operating.extent.depth = pop_size;
  p_influence_operating.kind = hipMemcpyHostToDevice;

  hipMemcpy3D(&p_influence_operating);
  return max_prot;
}


int transfert_data_to_gpu_thrust(
    int nb_gen,
    int multiply_population,
     int nb_signal, int degradationstep,
     int degradation_rate, double hill_shape_n, double hill_shape,
    int pop_size, int lifestep,
                          std::vector<std::vector<double>*> const &protein_concentration_list,
                          std::vector<std::vector<double>*> const &rna_basal_concentration_list,
                          std::vector<std::vector<std::vector<int>*>*> const &rna_produce_protein_list,
                          std::vector<std::vector<std::vector<double>*>*> const &rna_influence_enhancing_coef_list,
                          std::vector<std::vector<std::vector<double>*>*> const &rna_influence_operating_coef_list,
                                                    std::vector<std::vector<double>*> const &env_concentration_list) {

  int max_prot = 0, max_rna = 0;

  thrust::device_vector<double> gpu_thrust_protein_concentration;
  thrust::device_vector<double> gpu_thrust_rna_basal_concentration;
  thrust::device_vector<int>    gpu_thrust_rna_produce_protein;
  thrust::device_vector<double> gpu_thrust_rna_influence_enhancing_coef;
  thrust::device_vector<double> gpu_thrust_rna_influence_operating_coef;

  thrust::device_vector<double> gpu_thrust_environment_concentration;


  thrust::device_vector<int>    gpu_thrust_nb_protein;
  thrust::device_vector<int>    gpu_thrust_nb_rna_produce_protein;
  thrust::device_vector<int>    gpu_thrust_nb_influence;

  int g_max_rna;
  int g_max_protein;

  double* r_gpu_thrust_protein_concentration;
  double* r_gpu_thrust_rna_basal_concentration;
  int*    r_gpu_thrust_rna_produce_protein;
  double* r_gpu_thrust_rna_influence_enhancing_coef;
  double* r_gpu_thrust_rna_influence_operating_coef;

  double* r_gpu_thrust_environment_concentration;


  int* r_gpu_thrust_nb_protein;
  int* r_gpu_thrust_nb_rna_produce_protein;
  int* r_gpu_thrust_nb_influence;

  for (int i = 0; i < pop_size; i++) {
    max_prot = protein_concentration_list[i]->size() > max_prot ?
               protein_concentration_list[i]->size() : max_prot;
    max_rna = rna_basal_concentration_list[i]->size() > max_rna ?
              rna_basal_concentration_list[i]->size() : max_rna;
  }

  g_max_protein = max_prot;
  g_max_rna = max_rna;

  thrust::host_vector<double> thrust_protein_concentration(max_prot*pop_size);
  thrust::host_vector<double> thrust_rna_basal_concentration(max_rna*pop_size);
  thrust::host_vector<int> thrust_rna_produce_protein(max_rna*max_prot*pop_size);
  thrust::host_vector<double> thrust_rna_influence_enhancing_coef(max_rna*max_prot*pop_size);
  thrust::host_vector<double> thrust_rna_influence_operating_coef(max_rna*max_prot*pop_size);

  thrust::host_vector<int> thrust_nb_protein(pop_size);
  thrust::host_vector<int> thrust_nb_rna_produce_protein(max_prot*pop_size);
  thrust::host_vector<int> thrust_nb_influence(max_rna*pop_size);

  for (int i = 0; i < pop_size; i++){

    thrust_nb_protein[i] = protein_concentration_list[i]->size();


    thrust::copy(
                                        protein_concentration_list[i]->begin(),
                                        protein_concentration_list[i]->end(),
                                        thrust_protein_concentration.begin()+max_prot*i
                                        );

    thrust::copy(
                                        rna_basal_concentration_list[i]->begin(),
                                        rna_basal_concentration_list[i]->end(),
                                          thrust_rna_basal_concentration.begin()+max_rna*i
    );

    for (int prot_id = 0; prot_id < rna_produce_protein_list[i]->size(); prot_id++) {
      thrust_nb_rna_produce_protein[i*max_prot+prot_id] = rna_produce_protein_list[i]->at(prot_id)->size();
      thrust::copy(
                                          rna_produce_protein_list[i]->at(prot_id)->begin(),
                                          rna_produce_protein_list[i]->at(prot_id)->end(),
                                          thrust_rna_produce_protein.begin()+max_prot*max_rna*i+max_rna*prot_id
      );
    }

    for (int rna_id = 0; rna_id < rna_influence_enhancing_coef_list[i]->size(); rna_id++) {
      thrust_nb_influence[i*max_rna+rna_id] = rna_influence_enhancing_coef_list[i]->at(rna_id)->size();
      thrust::copy(
                                          rna_influence_enhancing_coef_list[i]->at(rna_id)->begin(),
                                          rna_influence_enhancing_coef_list[i]->at(rna_id)->end(),
                                          thrust_rna_influence_enhancing_coef.begin()+
                                                     max_prot*max_rna*i+max_prot*rna_id
      );
    }

    for (int rna_id = 0; rna_id < rna_influence_operating_coef_list[i]->size(); rna_id++) {
      thrust::copy(
                                                 rna_influence_operating_coef_list[i]->at(rna_id)->begin(),
                                                 rna_influence_operating_coef_list[i]->at(rna_id)->end(),
                                                 thrust_rna_influence_operating_coef.begin()+
                                                     max_prot*max_rna*i+max_prot*rna_id
      );

    }
  }

  thrust::host_vector<double> thrust_environment_concentration(lifestep*nb_signal);

  for (int i = 0; i < lifestep; i++) {
    thrust::copy(
                                            env_concentration_list[i]->begin(),
                                            env_concentration_list[i]->end(),
                                            thrust_environment_concentration.begin()+
                                               nb_signal*i
    );
  }

  printf("Copying to device");

  gpu_thrust_protein_concentration = thrust_protein_concentration;
  gpu_thrust_rna_basal_concentration = thrust_rna_basal_concentration;
  gpu_thrust_rna_produce_protein = thrust_rna_produce_protein;
  gpu_thrust_rna_influence_enhancing_coef = thrust_rna_influence_enhancing_coef;
  gpu_thrust_rna_influence_operating_coef = thrust_rna_influence_operating_coef;

  gpu_thrust_environment_concentration = thrust_environment_concentration;


  gpu_thrust_nb_protein = thrust_nb_protein;
  gpu_thrust_nb_rna_produce_protein = thrust_nb_rna_produce_protein;
  gpu_thrust_nb_influence = thrust_nb_influence;

  r_gpu_thrust_protein_concentration = thrust::raw_pointer_cast(gpu_thrust_protein_concentration.data());
  r_gpu_thrust_rna_basal_concentration = thrust::raw_pointer_cast(gpu_thrust_rna_basal_concentration.data());
  r_gpu_thrust_rna_produce_protein = thrust::raw_pointer_cast(gpu_thrust_rna_produce_protein.data());
  r_gpu_thrust_rna_influence_enhancing_coef = thrust::raw_pointer_cast(gpu_thrust_rna_influence_enhancing_coef.data());
  r_gpu_thrust_rna_influence_operating_coef = thrust::raw_pointer_cast(gpu_thrust_rna_influence_operating_coef.data());

  r_gpu_thrust_environment_concentration = thrust::raw_pointer_cast(gpu_thrust_environment_concentration.data());


  r_gpu_thrust_nb_protein = thrust::raw_pointer_cast(gpu_thrust_nb_protein.data());
  r_gpu_thrust_nb_rna_produce_protein = thrust::raw_pointer_cast(gpu_thrust_nb_rna_produce_protein.data());
  r_gpu_thrust_nb_influence = thrust::raw_pointer_cast(gpu_thrust_nb_influence.data());

  printf("Launching kernel\n");


  for (int gen = 0; gen < nb_gen; gen++) {

    process_delta_thrust << < 1024 * multiply_population, g_max_protein >> >
                                                          (nb_signal, degradationstep, degradation_rate,
                                                              g_max_protein, g_max_rna,
                                                              r_gpu_thrust_nb_rna_produce_protein, r_gpu_thrust_rna_produce_protein,
                                                              r_gpu_thrust_protein_concentration,
                                                              r_gpu_thrust_rna_basal_concentration, r_gpu_thrust_nb_protein,
                                                              r_gpu_thrust_rna_influence_enhancing_coef, r_gpu_thrust_rna_influence_operating_coef,
                                                              r_gpu_thrust_nb_influence,
                                                              r_gpu_thrust_environment_concentration, hill_shape, hill_shape_n);
  }
  return max_prot;
}


__global__
void process_delta(int nb_signal, int degradstep, int degradrate, int ***rna_produce_protein_array,
                   int **nb_rna_produce_protein, int *nb_rna_produce,   double **protein_concentration_array,
                   double **rna_basal_concentration_array, int *nb_protein_array, int *nb_rna_array,
                   double ***rna_influence_enhancing_coef_array, double ***rna_influence_operating_coef_array,
                   int **nb_rna_influence_enhancing_coef, int  **nb_rna_influence_operating_coef,
                   double **env_concentration_array, double hill_shape, double hill_shape_n) {



  double delta = 0;

  int indiv_id = blockIdx.x;
  int prot_id = threadIdx.x;

  if (prot_id < nb_protein_array[indiv_id] - nb_signal) {
    for (int j = 0; j < degradstep; j++) {
      for (int j = 0;
           j < nb_rna_produce_protein[indiv_id][prot_id]; j++) {
        double enhancer_activity = 0;
        double operator_activity = 0;

        int rna_id = rna_produce_protein_array[indiv_id][prot_id][j];

        for (int i = 0; i <
                        nb_rna_influence_enhancing_coef[indiv_id][rna_id]; i++) {

          enhancer_activity +=
              rna_influence_enhancing_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
          operator_activity +=
              rna_influence_operating_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
        }

        double enhancer_activity_pow_n = enhancer_activity == 0 ? 0 :
                                         powf(enhancer_activity, hill_shape_n);
        double operator_activity_pow_n = operator_activity == 0 ? 0 :
                                         powf(operator_activity, hill_shape_n);
        delta += rna_basal_concentration_array[indiv_id][rna_id]
                 * (hill_shape
                    / (operator_activity_pow_n + hill_shape))
                 * (1 +
                    ((1 / rna_basal_concentration_array[indiv_id][rna_id]
                     ) -
                     1)
                    * (enhancer_activity_pow_n /
                       (enhancer_activity_pow_n + hill_shape)));
      }

      delta -=
          degradrate *
          protein_concentration_array[indiv_id][prot_id];
      delta *= 1 / (double) degradstep;

      __syncthreads();

      protein_concentration_array[indiv_id][prot_id] = delta;
    }
  }
}



__global__
void process_delta_float(int nb_signal, int degradstep, int degradrate, int ***rna_produce_protein_array,
                   int **nb_rna_produce_protein, int *nb_rna_produce,   float **protein_concentration_array,
                   float **rna_basal_concentration_array, int *nb_protein_array, int *nb_rna_array,
                   float ***rna_influence_enhancing_coef_array, float ***rna_influence_operating_coef_array,
                   int **nb_rna_influence_enhancing_coef, int  **nb_rna_influence_operating_coef,
                   float **env_concentration_array, float hill_shape, float hill_shape_n) {



  float delta = 0;

  int indiv_id = blockIdx.x;
  int prot_id = threadIdx.x;

  if (prot_id < nb_protein_array[indiv_id] - nb_signal) {
    for (int j = 0; j < degradstep; j++) {
      for (int j = 0;
           j < nb_rna_produce_protein[indiv_id][prot_id]; j++) {
        float enhancer_activity = 0;
        float operator_activity = 0;

        int rna_id = rna_produce_protein_array[indiv_id][prot_id][j];

        for (int i = 0; i <
                        nb_rna_influence_enhancing_coef[indiv_id][rna_id]; i++) {

          enhancer_activity +=
              rna_influence_enhancing_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
          operator_activity +=
              rna_influence_operating_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
        }

        float enhancer_activity_pow_n = enhancer_activity == 0 ? 0 :
                                         powf(enhancer_activity, hill_shape_n);
        float operator_activity_pow_n = operator_activity == 0 ? 0 :
                                         powf(operator_activity, hill_shape_n);
        delta += rna_basal_concentration_array[indiv_id][rna_id]
                 * (hill_shape
                    / (operator_activity_pow_n + hill_shape))
                 * (1 +
                    ((1 / rna_basal_concentration_array[indiv_id][rna_id]
                     ) -
                     1)
                    * (enhancer_activity_pow_n /
                       (enhancer_activity_pow_n + hill_shape)));
      }

      delta -=
          degradrate *
          protein_concentration_array[indiv_id][prot_id];
      delta *= 1 / (float) degradstep;

      __syncthreads();

      protein_concentration_array[indiv_id][prot_id] = delta;
    }
  }
}


__global__
void process_delta_dense(int nb_signal, int degradstep, int degradrate, int ***rna_produce_protein_array,
                   int **nb_rna_produce_protein, int *nb_rna_produce,   double **protein_concentration_array,
                   double **rna_basal_concentration_array, int *nb_protein_array, int *nb_rna_array,
                   double ***rna_influence_enhancing_coef_array, double ***rna_influence_operating_coef_array,
                   int **nb_rna_influence_enhancing_coef, int  **nb_rna_influence_operating_coef,
                   double **env_concentration_array, double hill_shape, double hill_shape_n) {



  double delta = 0;

  int indiv_id = blockIdx.x;
  int prot_id = threadIdx.x;

  if (prot_id < nb_protein_array[indiv_id] - nb_signal) {
    for (int j = 0; j < degradstep; j++) {
      for (int j = 0;
           j < nb_rna_produce_protein[indiv_id][prot_id]; j++) {
        double enhancer_activity = 0;
        double operator_activity = 0;

        int rna_id = rna_produce_protein_array[indiv_id][prot_id][j];

        for (int i = 0; i <
                        nb_rna_influence_enhancing_coef[indiv_id][rna_id]; i++) {

          enhancer_activity +=
              rna_influence_enhancing_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
          operator_activity +=
              rna_influence_operating_coef_array[indiv_id][rna_id][i]
              * protein_concentration_array[indiv_id][i];
        }

        double enhancer_activity_pow_n = enhancer_activity == 0 ? 0 :
                                         powf(enhancer_activity, hill_shape_n);
        double operator_activity_pow_n = operator_activity == 0 ? 0 :
                                         powf(operator_activity, hill_shape_n);
        delta += rna_basal_concentration_array[indiv_id][rna_id]
                 * (hill_shape
                    / (operator_activity_pow_n + hill_shape))
                 * (1 +
                    ((1 / rna_basal_concentration_array[indiv_id][rna_id]
                     ) -
                     1)
                    * (enhancer_activity_pow_n /
                       (enhancer_activity_pow_n + hill_shape)));
      }

      delta -=
          degradrate *
          protein_concentration_array[indiv_id][prot_id];
      delta *= 1 / (double) degradstep;

      __syncthreads();

      protein_concentration_array[indiv_id][prot_id] = delta;
    }
  }
}


__global__
void process_delta_thrust(int nb_signal, int degradstep, int degradrate,
                          int max_prot, int max_rna,
                   int *nb_rna_produce_protein, int *rna_produce_protein,
                   double *protein_concentration_array,
                   double *rna_basal_concentration_array, int *nb_protein_array,
                   double *rna_influence_enhancing_coef_array, double *rna_influence_operating_coef_array,
                   int *nb_rna_influence_enhancing_coef,
                   double *env_concentration_array, double hill_shape, double hill_shape_n) {



  double delta = 0;

  int indiv_id = blockIdx.x;
  int prot_id = threadIdx.x;

  if (prot_id < nb_protein_array[indiv_id] - nb_signal) {
    for (int j = 0; j < degradstep; j++) {
      for (int j = 0;
           j < nb_rna_produce_protein[indiv_id*max_prot+prot_id]; j++) {
        double enhancer_activity = 0;
        double operator_activity = 0;

        int rna_id = rna_produce_protein[indiv_id*max_prot*max_rna+prot_id*max_rna+j];

        for (int i = 0; i <
                        nb_rna_influence_enhancing_coef[indiv_id*max_rna+rna_id]; i++) {

          enhancer_activity +=
              rna_influence_enhancing_coef_array[indiv_id*max_rna*max_prot+rna_id*max_prot+i]
              * protein_concentration_array[indiv_id*max_prot+i];
          operator_activity +=
              rna_influence_operating_coef_array[indiv_id*max_rna*max_prot+rna_id*max_prot+i]
              * protein_concentration_array[indiv_id*max_prot+i];
        }

        double enhancer_activity_pow_n = enhancer_activity == 0 ? 0 :
                                         powf(enhancer_activity, hill_shape_n);
        double operator_activity_pow_n = operator_activity == 0 ? 0 :
                                         powf(operator_activity, hill_shape_n);
        delta += rna_basal_concentration_array[indiv_id*max_rna+rna_id]
                 * (hill_shape
                    / (operator_activity_pow_n + hill_shape))
                 * (1 +
                    ((1 / rna_basal_concentration_array[indiv_id*max_rna+rna_id]
                     ) -
                     1)
                    * (enhancer_activity_pow_n /
                       (enhancer_activity_pow_n + hill_shape)));
      }

      delta -=
          degradrate *
          protein_concentration_array[indiv_id*max_prot+prot_id];
      delta *= 1 / (double) degradstep;

      __syncthreads();

      protein_concentration_array[indiv_id*max_prot+prot_id] = delta;
    }
  }
}
